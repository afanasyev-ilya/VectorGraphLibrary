#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define __USE_GPU__

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define INT_ELEMENTS_PER_EDGE 5.0

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "graph_library.h"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char * argv[])
{
    try
    {
        cout << "SSSP (Single Source Shortest Paths) test..." << endl;

        // parse args
        Parser parser;
        parser.parse_args(argc, argv);

        select_device(parser.get_device_num());

        VectCSRGraph graph;
        if(parser.get_compute_mode() == GENERATE_NEW_GRAPH)
        {
            EdgesListGraph el_graph;
            int v = pow(2.0, parser.get_scale());
            if(parser.get_graph_type() == RMAT)
                GraphGenerationAPI::R_MAT(el_graph, v, v * parser.get_avg_degree(), 57, 19, 19, 5, DIRECTED_GRAPH);
            else if(parser.get_graph_type() == RANDOM_UNIFORM)
                GraphGenerationAPI::random_uniform(el_graph, v, v * parser.get_avg_degree(), DIRECTED_GRAPH);
            graph.import(el_graph);
        }
        else if(parser.get_compute_mode() == LOAD_GRAPH_FROM_FILE)
        {
            Timer tm;
            tm.start();
            if(!graph.load_from_binary_file(parser.get_graph_file_name()))
                throw "Error: graph file not found";
            tm.end();
            tm.print_time_stats("Graph load");
        }

        // print size of VectCSR graph
        graph.print_size();

        // move graph to device for better performance
        graph.move_to_device();

        cout << "Computations started..." << endl;
        cout << "Doing " << parser.get_number_of_rounds() << " SSSP iterations..." << endl;

        // do calculations
        cout << "Computations started..." << endl;
        cout << "Doing " << parser.get_number_of_rounds() << " SSSP iterations..." << endl;
        EdgesArray_Vect<float> capacities(graph);
        capacities.set_all_random(MAX_WEIGHT);
        for(int i = 0; i < parser.get_number_of_rounds(); i++)
        {
            int source_vertex = graph.select_random_vertex(ORIGINAL);
            VerticesArray<float> widths(graph, SCATTER);

            performance_stats.reset_timers();
            SSWP::vgl_dijkstra(graph, capacities, widths, source_vertex);
            performance_stats.update_timer_stats();
            performance_stats.print_timers_stats();

            // check if required
            if(parser.get_check_flag())
            {
                graph.move_to_host();
                widths.move_to_host();
                capacities.move_to_host();

                VerticesArray<float> check_widths(graph, SCATTER);
                SSWP::seq_dijkstra(graph, capacities, check_widths, source_vertex);
                verify_results(widths, check_widths, 20);

                graph.move_to_device();
                widths.move_to_device();
                capacities.move_to_device();
            }
        }
        performance_stats.print_perf(graph.get_edges_count());
    }
    catch (string error)
    {
        cout << error << endl;
    }
    catch (const char * error)
    {
        cout << error << endl;
    }
    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
