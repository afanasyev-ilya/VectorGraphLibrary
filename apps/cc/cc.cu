#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define __USE_GPU__

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define INT_ELEMENTS_PER_EDGE 4.0

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "graph_library.h"

////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    try
    {
        cout << "CC (Connected Components) test..." << endl;

        // parse args
        Parser parser;
        parser.parse_args(argc, argv);

        select_device(parser.get_device_num());

        VectCSRGraph graph;
        if(parser.get_compute_mode() == GENERATE_NEW_GRAPH)
        {
            EdgesListGraph el_graph;
            int v = pow(2.0, parser.get_scale());
            if(parser.get_graph_type() == RMAT)
                GraphGenerationAPI::R_MAT(el_graph, v, v * parser.get_avg_degree(), 57, 19, 19, 5, DIRECTED_GRAPH);
            else if(parser.get_graph_type() == RANDOM_UNIFORM)
                GraphGenerationAPI::random_uniform(el_graph, v, v * parser.get_avg_degree(), DIRECTED_GRAPH);
            graph.import(el_graph);
        }
        else if(parser.get_compute_mode() == LOAD_GRAPH_FROM_FILE)
        {
            Timer tm;
            tm.start();
            if(!graph.load_from_binary_file(parser.get_graph_file_name()))
                throw "Error: graph file not found";
            tm.end();
            tm.print_time_stats("Graph load");
        }

        // print size of VectCSR graph
        graph.print_size();

        // move graph to device for better performance
        graph.move_to_device();

        // do calculations
        cout << "Computations started..." << endl;
        cout << "Running CC algorithm " << parser.get_number_of_rounds() << " times..." << endl;
        for(int i = 0; i < parser.get_number_of_rounds(); i++)
        {
            VerticesArray<int> components(graph, SCATTER);
            performance_stats.reset_timers();
            ConnectedComponents::gpu_shiloach_vishkin(graph, components);
            performance_stats.update_timer_stats();
            performance_stats.print_timers_stats();

            // check correctness
            if(parser.get_check_flag())
            {
                VerticesArray<int> check_components(graph, SCATTER);
                ConnectedComponents::seq_bfs_based(graph, check_components);
                equal_components(components, check_components);
            }
        }

        performance_stats.print_perf(graph.get_edges_count());
    }
    catch (string error)
    {
        cout << error << endl;
    }
    catch (const char * error)
    {
        cout << error << endl;
    }
    return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
