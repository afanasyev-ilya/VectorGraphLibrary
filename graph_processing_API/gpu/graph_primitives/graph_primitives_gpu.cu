#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphPrimitivesGPU::GraphPrimitivesGPU()
{
    hipStreamCreate(&grid_processing_stream);
    hipStreamCreate(&block_processing_stream);
    hipStreamCreate(&warp_processing_stream);
    hipStreamCreate(&vwp_16_processing_stream);
    hipStreamCreate(&vwp_8_processing_stream);
    hipStreamCreate(&vwp_4_processing_stream);
    hipStreamCreate(&vwp_2_processing_stream);
    hipStreamCreate(&thread_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphPrimitivesGPU::~GraphPrimitivesGPU()
{
    hipStreamDestroy(block_processing_stream);
    hipStreamDestroy(warp_processing_stream);
    hipStreamDestroy(thread_processing_stream);
    hipStreamDestroy(grid_processing_stream);
    hipStreamDestroy(vwp_16_processing_stream);
    hipStreamDestroy(vwp_8_processing_stream);
    hipStreamDestroy(vwp_4_processing_stream);
    hipStreamDestroy(vwp_2_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
