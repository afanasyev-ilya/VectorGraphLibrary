#include "hip/hip_runtime.h"
#pragma once

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

struct is_active
{
    __device__
    bool operator()(const int x)
    {
        return x != -1;
    }
};

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

struct is_not_active
{
    __device__
    bool operator()(const int x)
    {
        return x == -1;
    }
};

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Condition>
void __global__ copy_frontier_ids_kernel(int *_frontier_ids,
                                         int *_frontier_flags,
                                         const int _vertices_count,
                                         Condition cond)
{
    register const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < _vertices_count)
    {
        if(cond(idx) == true)
        {
            _frontier_ids[idx] = idx;
            _frontier_flags[idx] = IN_FRONTIER_FLAG;
        }
        else
        {
            _frontier_ids[idx] = -1;
            _frontier_flags[idx] = NOT_IN_FRONTIER_FLAG;
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename _TVertexValue, typename _TEdgeWeight, typename Condition>
void GraphPrimitivesGPU::generate_new_frontier(ExtendedCSRGraph<_TVertexValue, _TEdgeWeight> &_graph,
                                               FrontierGPU &_frontier,
                                               Condition &&cond)
{
    int vertices_count = _graph.get_vertices_count();
    _frontier.type = SPARSE_FRONTIER;


    SAFE_KERNEL_CALL((copy_frontier_ids_kernel<<<(vertices_count-1)/BLOCK_SIZE+1, BLOCK_SIZE>>>(_frontier.ids, _frontier.flags,
                                                                                                vertices_count, cond)));
    int *new_end = thrust::remove_if(thrust::device, _frontier.ids, _frontier.ids + vertices_count, is_not_active());
    _frontier.current_size = new_end - _frontier.ids;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////