#include "hip/hip_runtime.h"
#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <class EdgeOperation>
void __global__ grid_per_vertex_kernel_child(const long long *_vertex_pointers,
                                             const int *_adjacent_ids,
                                             const int _vertices_count,
                                             const int _src_id,
                                             const int _connections_count,
                                             EdgeOperation edge_op,
                                             int _frontier_pos,
                                             int *_new_frontier_ids,
                                             bool _generate_frontier,
                                             int *_new_frontier_size)
{
    const int src_id = _src_id;
    const long long edge_start = _vertex_pointers[src_id];
    const long long edge_pos = blockIdx.x * blockDim.x + threadIdx.x;

    if(edge_pos < _connections_count)
    {
        const long long int global_edge_pos = edge_start + edge_pos;
        const int dst_id = _adjacent_ids[global_edge_pos];
        const int local_edge_pos = edge_pos;
        edge_op(src_id, dst_id, local_edge_pos, global_edge_pos, _frontier_pos);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <class EdgeOperation, class VertexPreprocessOperation, class VertexPostprocessOperation>
void __global__ grid_per_vertex_kernel(const long long *_vertex_pointers,
                                       const int *_adjacent_ids,
                                       const int *_frontier_ids,
                                       const int _vertices_count,
                                       const int _vertex_part_start,
                                       const int _vertex_part_end,
                                       EdgeOperation edge_op,
                                       VertexPreprocessOperation vertex_preprocess_op,
                                       VertexPostprocessOperation vertex_postprocess_op,
                                       int *_new_frontier_ids,
                                       bool _generate_frontier,
                                       int *_new_frontier_size)
{
    const int frontier_pos = blockIdx.x * blockDim.x + threadIdx.x + _vertex_part_start;
    if(frontier_pos < _vertex_part_end)
    {
        const int src_id = _frontier_ids[frontier_pos];
        const int connections_count = _vertex_pointers[src_id + 1] - _vertex_pointers[src_id];

        vertex_preprocess_op(src_id, frontier_pos, connections_count);

        dim3 child_threads(BLOCK_SIZE);
        dim3 child_blocks((connections_count - 1) / BLOCK_SIZE + 1);
        grid_per_vertex_kernel_child <<< child_blocks, child_threads >>> (_vertex_pointers, _adjacent_ids,
                _vertices_count, src_id, connections_count, edge_op, frontier_pos, _new_frontier_ids, _generate_frontier, _new_frontier_size);

        vertex_postprocess_op(src_id, frontier_pos, connections_count);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <class EdgeOperation, class VertexPreprocessOperation, class VertexPostprocessOperation>
void __global__ block_per_vertex_kernel(const long long *_vertex_pointers,
                                        const int *_adjacent_ids,
                                        const int *_frontier_ids,
                                        const int _vertices_count,
                                        const int _vertex_part_start,
                                        const int _vertex_part_end,
                                        EdgeOperation edge_op,
                                        VertexPreprocessOperation vertex_preprocess_op,
                                        VertexPostprocessOperation vertex_postprocess_op,
                                        int *_new_frontier_ids,
                                        bool _generate_frontier,
                                        int *_new_frontier_size)
{
    const int frontier_pos = blockIdx.x + _vertex_part_start;
    if(frontier_pos < _vertex_part_end)
    {
        const int src_id = _frontier_ids[frontier_pos];
        const long long edge_start = _vertex_pointers[src_id];
        const int connections_count =  _vertex_pointers[src_id + 1] - _vertex_pointers[src_id];
        vertex_preprocess_op(src_id, frontier_pos, connections_count);

        for(register int edge_pos = threadIdx.x; edge_pos < connections_count; edge_pos += BLOCK_SIZE)
        {
            if(edge_pos < connections_count)
            {
                const long long int global_edge_pos = edge_start + edge_pos;
                const int dst_id = _adjacent_ids[global_edge_pos];
                const int local_edge_pos = edge_pos;
                edge_op(src_id, dst_id, local_edge_pos, global_edge_pos, frontier_pos);
            }
        }

        vertex_postprocess_op(src_id, frontier_pos, connections_count);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <class EdgeOperation, class VertexPreprocessOperation, class VertexPostprocessOperation>
void __global__ warp_per_vertex_kernel(const long long *_vertex_pointers,
                                       const int *_adjacent_ids,
                                       const int *_frontier_ids,
                                       const int _vertices_count,
                                       const int _vertex_part_start,
                                       const int _vertex_part_end,
                                       EdgeOperation edge_op,
                                       VertexPreprocessOperation vertex_preprocess_op,
                                       VertexPostprocessOperation vertex_postprocess_op,
                                       int *_new_frontier_ids,
                                       bool _generate_frontier,
                                       int *_new_frontier_size)
{
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int frontier_pos = blockIdx.x * (blockDim.x/ WARP_SIZE) + warp_id + _vertex_part_start;

    if(frontier_pos < _vertex_part_end)
    {
        const int src_id = _frontier_ids[frontier_pos];
        const long long edge_start = _vertex_pointers[src_id];
        const int connections_count = _vertex_pointers[src_id + 1] - _vertex_pointers[src_id];
        vertex_preprocess_op(src_id, frontier_pos, connections_count);

        for(register int edge_pos = lane_id; edge_pos < connections_count; edge_pos += WARP_SIZE)
        {
            if(edge_pos < connections_count)
            {
                const long long int global_edge_pos = edge_start + edge_pos;
                const int dst_id = _adjacent_ids[global_edge_pos];
                const int local_edge_pos = edge_pos;
                edge_op(src_id, dst_id, local_edge_pos, global_edge_pos, frontier_pos);
            }
        }

        vertex_postprocess_op(src_id, frontier_pos, connections_count);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename EdgeOperation, typename VertexPreprocessOperation, typename VertexPostprocessOperation>
void __global__ thread_per_vertex_kernel(const long long *_vertex_pointers,
                                         const int *_adjacent_ids,
                                         const int *_frontier_ids,
                                         const int _vertices_count,
                                         const int _vertex_part_start,
                                         const int _vertex_part_end,
                                         EdgeOperation edge_op,
                                         VertexPreprocessOperation vertex_preprocess_op,
                                         VertexPostprocessOperation vertex_postprocess_op,
                                         int *_new_frontier_ids,
                                         bool _generate_frontier,
                                         int *_new_frontier_size)
{
    const int frontier_pos = blockIdx.x * blockDim.x + threadIdx.x + _vertex_part_start;

    if(frontier_pos < _vertex_part_end)
    {
        const int src_id = _frontier_ids[frontier_pos];

        const long long edge_start = _vertex_pointers[src_id];
        const int connections_count = _vertex_pointers[src_id + 1] - _vertex_pointers[src_id];

        vertex_preprocess_op(src_id, frontier_pos, connections_count);

        for(register int edge_pos = 0; edge_pos < connections_count; edge_pos++)
        {
            if(edge_pos < connections_count)
            {
                const long long int global_edge_pos = edge_start + edge_pos;
                const int dst_id = _adjacent_ids[global_edge_pos];
                const int local_edge_pos = edge_pos;
                edge_op(src_id, dst_id, local_edge_pos, global_edge_pos, frontier_pos);
            }
        }

        vertex_postprocess_op(src_id, frontier_pos, connections_count);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <int VirtualWarpSize, typename EdgeOperation, typename VertexPreprocessOperation, typename VertexPostprocessOperation>
void __global__ virtual_warp_per_vertex_kernel(const long long *_vertex_pointers,
                                               const int *_adjacent_ids,
                                               const int *_frontier_ids,
                                               const int _vertices_count,
                                               const int _vertex_part_start,
                                               const int _vertex_part_end,
                                               EdgeOperation edge_op,
                                               VertexPreprocessOperation vertex_preprocess_op,
                                               VertexPostprocessOperation vertex_postprocess_op,
                                               int *_new_frontier_ids,
                                               bool _generate_frontier,
                                               int *_new_frontier_size)
{
    const int virtual_warp_id = threadIdx.x / VirtualWarpSize;
    const int position_in_virtual_warp = threadIdx.x % VirtualWarpSize;

    const int frontier_pos = blockIdx.x * (blockDim.x / VirtualWarpSize) + virtual_warp_id + _vertex_part_start;

    if(frontier_pos < _vertex_part_end)
    {
        const int src_id = _frontier_ids[frontier_pos];

        const long long edge_start = _vertex_pointers[src_id];
        const int connections_count = _vertex_pointers[src_id + 1] - _vertex_pointers[src_id];

        vertex_preprocess_op(src_id, frontier_pos, connections_count);

        for(register int edge_pos = position_in_virtual_warp; edge_pos < connections_count; edge_pos += VirtualWarpSize)
        {
            if(edge_pos < connections_count)
            {
                const long long int global_edge_pos = edge_start + edge_pos;
                const int dst_id = _adjacent_ids[global_edge_pos];
                const int local_edge_pos = edge_pos;
                edge_op(src_id, dst_id, local_edge_pos, global_edge_pos, frontier_pos);
            }
        }

        vertex_postprocess_op(src_id, frontier_pos, connections_count);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename _TVertexValue, typename _TEdgeWeight, typename EdgeOperation, typename VertexPreprocessOperation,
        typename VertexPostprocessOperation>
void GraphPrimitivesGPU::advance_sparse(UndirectedGraph &_graph,
                                        FrontierGPU &_frontier,
                                        EdgeOperation edge_op,
                                        VertexPreprocessOperation vertex_preprocess_op,
                                        VertexPostprocessOperation vertex_postprocess_op,
                                        bool _generate_frontier)
{
    #ifdef __PRINT_API_PERFORMANCE_STATS__
    hipDeviceSynchronize();
    double t1 = omp_get_wtime();
    #endif
    LOAD_UNDIRECTED_CSR_GRAPH_DATA(_graph);

    int grid_threshold_start = 0;
    int grid_threshold_end = 0;
    int block_threshold_start = 0;
    int block_threshold_end = 0;
    int warp_threshold_start = 0;
    int warp_threshold_end = 0;

    int vwp_16_threshold_start = 0;
    int vwp_16_threshold_end = 0;
    int vwp_8_threshold_start = 0;
    int vwp_8_threshold_end = 0;
    int vwp_4_threshold_start = 0;
    int vwp_4_threshold_end = 0;
    int vwp_2_threshold_start = 0;
    int vwp_2_threshold_end = 0;

    int thread_threshold_start = 0;
    int thread_threshold_end = 0;

    _frontier.split_sorted_frontier(vertex_pointers, grid_threshold_start, grid_threshold_end, block_threshold_start,
                                    block_threshold_end, warp_threshold_start, warp_threshold_end,
                                    vwp_16_threshold_start, vwp_16_threshold_end,
                                    vwp_8_threshold_start, vwp_8_threshold_end,
                                    vwp_4_threshold_start, vwp_4_threshold_end,
                                    vwp_2_threshold_start, vwp_2_threshold_end,
                                    thread_threshold_start, thread_threshold_end);


    int *tmp_new_frontier_buffer = _frontier.flags;
    int *new_frontier_size;
    if(_generate_frontier)
        MemoryAPI::allocate_managed_array(&new_frontier_size, 1);

    int grid_vertices_count = grid_threshold_end - grid_threshold_start;
    if (grid_vertices_count > 0)
    {
        grid_per_vertex_kernel <<< grid_vertices_count, 1, 0, grid_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, grid_threshold_start,
                 grid_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int block_vertices_count = block_threshold_end - block_threshold_start;
    if (block_vertices_count > 0)
    {
        block_per_vertex_kernel <<< block_vertices_count, BLOCK_SIZE, 0, block_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, block_threshold_start,
                 block_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int warp_vertices_count = warp_threshold_end - warp_threshold_start;
    if (warp_vertices_count > 0)
    {
        warp_per_vertex_kernel <<< WARP_SIZE*(warp_vertices_count - 1)/BLOCK_SIZE + 1, BLOCK_SIZE, 0, warp_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, warp_threshold_start,
                 warp_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int vwp_16_vertices_count = vwp_16_threshold_end - vwp_16_threshold_start;
    if(vwp_16_vertices_count > 0)
    {
        virtual_warp_per_vertex_kernel<16> <<< 16*(vwp_16_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, vwp_16_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, vwp_16_threshold_start,
                 vwp_16_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int vwp_8_vertices_count = vwp_8_threshold_end - vwp_8_threshold_start;
    if(vwp_8_vertices_count > 0)
    {
        virtual_warp_per_vertex_kernel<8> <<< 8*(vwp_8_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, vwp_8_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, vwp_8_threshold_start,
                 vwp_8_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int vwp_4_vertices_count = vwp_4_threshold_end - vwp_4_threshold_start;
    if(vwp_4_vertices_count > 0)
    {
        virtual_warp_per_vertex_kernel<4> <<< 4*(vwp_4_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, vwp_4_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, vwp_4_threshold_start,
                 vwp_4_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int vwp_2_vertices_count = vwp_2_threshold_end - vwp_2_threshold_start;
    if(vwp_2_vertices_count > 0)
    {
        virtual_warp_per_vertex_kernel<2> <<< 2*(vwp_2_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, vwp_2_processing_stream >>>
                (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, vwp_2_threshold_start,
                 vwp_2_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }

    int thread_vertices_count = thread_threshold_end - thread_threshold_start;
    if (thread_vertices_count > 0)
    {
        thread_per_vertex_kernel <<< (thread_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, thread_processing_stream >>>
                                                                                                  (vertex_pointers, adjacent_ids, _frontier.ids, vertices_count, thread_threshold_start,
                                                                                                          thread_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op, tmp_new_frontier_buffer, _generate_frontier, new_frontier_size);
    }
    hipDeviceSynchronize();

    #ifdef __PRINT_API_PERFORMANCE_STATS__
    double t2 = omp_get_wtime();
    hipDeviceSynchronize();
    INNER_WALL_TIME += t2 - t1;
    INNER_ADVANCE_TIME += t2 - t1;
    int work = this->estimate_advance_work(_graph, _frontier);
    INNER_WALL_WORK += work;
    cout << "frontier size: " << _frontier.size() << "/" << vertices_count << ", " << 100.0*_frontier.size()/vertices_count << "%" << endl;
    cout << "advance time: " << (t2 - t1)*1000.0 << " ms" << endl;
    cout << "advance sparse BW: " << sizeof(int)*INT_ELEMENTS_PER_EDGE*work/((t2-t1)*1e9) << " GB/s" << endl << endl;
    #endif
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////