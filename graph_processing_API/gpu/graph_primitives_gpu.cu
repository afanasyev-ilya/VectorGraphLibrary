#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "traversal_kernels.cu"
#include "init_kernels.cu"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphPrimitivesGPU::GraphPrimitivesGPU()
{
    hipStreamCreate(&grid_processing_stream);
    hipStreamCreate(&block_processing_stream);
    hipStreamCreate(&warp_processing_stream);
    hipStreamCreate(&thread_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphPrimitivesGPU::~GraphPrimitivesGPU()
{
    hipStreamDestroy(block_processing_stream);
    hipStreamDestroy(warp_processing_stream);
    hipStreamDestroy(thread_processing_stream);
    hipStreamDestroy(grid_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename InitOperation>
void GraphPrimitivesGPU::init(int _size, InitOperation init_op)
{
    init_kernel <<< (_size - 1)/BLOCK_SIZE + 1, BLOCK_SIZE >>> (_size, init_op);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename _TVertexValue, typename _TEdgeWeight, typename EdgeOperation, typename VertexPreprocessOperation,
        typename VertexPostprocessOperation>
void GraphPrimitivesGPU::advance(ExtendedCSRGraph<_TVertexValue, _TEdgeWeight> &_graph,
                                 FrontierGPU &_frontier,
                                 EdgeOperation edge_op,
                                 VertexPreprocessOperation vertex_preprocess_op,
                                 VertexPostprocessOperation vertex_postprocess_op) {
    LOAD_EXTENDED_CSR_GRAPH_DATA(_graph);

    int grid_threshold_start = 0;
    int grid_threshold_end = 0;
    int block_threshold_start = 0;
    int block_threshold_end = 0;
    int warp_threshold_start = 0;
    int warp_threshold_end = 0;
    int thread_threshold_start = 0;
    int thread_threshold_end = 0;

    _frontier.split_sorted_frontier(outgoing_ptrs, grid_threshold_start, grid_threshold_end, block_threshold_start,
                                    block_threshold_end, warp_threshold_start, warp_threshold_end,
                                    thread_threshold_start, thread_threshold_end);

    int grid_vertices_count = grid_threshold_end - grid_threshold_start;
    if (grid_vertices_count > 0)
    {
        grid_per_vertex_kernel <<< grid_vertices_count, 1, 0, grid_processing_stream >>>
                (outgoing_ptrs, outgoing_ids, _frontier.frontier_ids, vertices_count, grid_threshold_start,
                 grid_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op);
    }

    int block_vertices_count = block_threshold_end - block_threshold_start;
    if (block_vertices_count > 0)
    {
        block_per_vertex_kernel <<< block_vertices_count, BLOCK_SIZE, 0, block_processing_stream >>>
               (outgoing_ptrs, outgoing_ids, _frontier.frontier_ids, vertices_count, block_threshold_start,
                block_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op);
    }

    int warp_vertices_count = warp_threshold_end - warp_threshold_start;
    if (warp_vertices_count > 0)
    {
        warp_per_vertex_kernel <<< warp_vertices_count, WARP_SIZE, 0, warp_processing_stream >>>
              (outgoing_ptrs, outgoing_ids, _frontier.frontier_ids, vertices_count, warp_threshold_start,
               warp_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op);
    }

    int thread_vertices_count = thread_threshold_end - thread_threshold_start;
    if (thread_vertices_count)
    {
        thread_per_vertex_kernel <<< (thread_vertices_count - 1) / BLOCK_SIZE + 1, BLOCK_SIZE, 0, thread_processing_stream >>>
                (outgoing_ptrs, outgoing_ids, _frontier.frontier_ids, vertices_count, thread_threshold_start,
                 thread_threshold_end, edge_op, vertex_preprocess_op, vertex_postprocess_op);
    }
    hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
