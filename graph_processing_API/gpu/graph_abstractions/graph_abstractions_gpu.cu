#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphAbstractionsGPU::GraphAbstractionsGPU(VectCSRGraph &_graph, TraversalDirection _initial_traversal)
{
    processed_graph_ptr = &_graph;
    current_traversal_direction = _initial_traversal;
    direction_shift = _graph.get_edges_count();

    hipStreamCreate(&grid_processing_stream);
    hipStreamCreate(&block_processing_stream);
    hipStreamCreate(&warp_processing_stream);
    hipStreamCreate(&vwp_16_processing_stream);
    hipStreamCreate(&vwp_8_processing_stream);
    hipStreamCreate(&vwp_4_processing_stream);
    hipStreamCreate(&vwp_2_processing_stream);
    hipStreamCreate(&thread_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphAbstractionsGPU::GraphAbstractionsGPU(ShardedCSRGraph &_graph, TraversalDirection _initial_traversal)
{
    processed_graph_ptr = NULL; // TODO
    current_traversal_direction = _initial_traversal;
    direction_shift = _graph.get_edges_count(); // TODO

    hipStreamCreate(&grid_processing_stream);
    hipStreamCreate(&block_processing_stream);
    hipStreamCreate(&warp_processing_stream);
    hipStreamCreate(&vwp_16_processing_stream);
    hipStreamCreate(&vwp_8_processing_stream);
    hipStreamCreate(&vwp_4_processing_stream);
    hipStreamCreate(&vwp_2_processing_stream);
    hipStreamCreate(&thread_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

GraphAbstractionsGPU::~GraphAbstractionsGPU()
{
    hipStreamDestroy(block_processing_stream);
    hipStreamDestroy(warp_processing_stream);
    hipStreamDestroy(thread_processing_stream);
    hipStreamDestroy(grid_processing_stream);
    hipStreamDestroy(vwp_16_processing_stream);
    hipStreamDestroy(vwp_8_processing_stream);
    hipStreamDestroy(vwp_4_processing_stream);
    hipStreamDestroy(vwp_2_processing_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
