#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "frontier_kernels.cu"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

FrontierGPU::FrontierGPU(int _vertices_count)
{
    max_size = _vertices_count;
    current_size = 0;
    hipMalloc((void**)&ids, max_size*sizeof(int));
    hipMalloc((void**)&flags, max_size*sizeof(int));
    hipMemset(ids, 0, max_size*sizeof(int));
    hipMemset(flags, 0, max_size*sizeof(int));
    type = ALL_ACTIVE_FRONTIER;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

FrontierGPU::~FrontierGPU()
{
    hipFree(ids);
    hipFree(flags);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void FrontierGPU::set_all_active()
{
    type = ALL_ACTIVE_FRONTIER;

    SAFE_KERNEL_CALL((set_all_active_frontier_kernel<<< max_size/BLOCK_SIZE, BLOCK_SIZE >>> (ids, flags, max_size)));
    current_size = max_size;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void FrontierGPU::split_sorted_frontier(const long long *_vertex_pointers,
                                        int &_grid_threshold_start,
                                        int &_grid_threshold_end,
                                        int &_block_threshold_start,
                                        int &_block_threshold_end,
                                        int &_warp_threshold_start,
                                        int &_warp_threshold_end,
                                        int &_thread_threshold_start,
                                        int &_thread_threshold_end)
{
    int *dev_grid_threshold_vertex;
    int *dev_block_threshold_vertex;
    int *dev_warp_threshold_vertex;
    hipMalloc((void**)&dev_grid_threshold_vertex, sizeof(int));
    hipMalloc((void**)&dev_block_threshold_vertex, sizeof(int));
    hipMalloc((void**)&dev_warp_threshold_vertex, sizeof(int));

    hipMemset(dev_grid_threshold_vertex, 0, sizeof(int));
    hipMemset(dev_block_threshold_vertex, 0, sizeof(int));
    hipMemset(dev_warp_threshold_vertex, 0, sizeof(int));

    split_frontier_kernel<<<(current_size - 1)/BLOCK_SIZE+1, BLOCK_SIZE>>>(_vertex_pointers, ids,
            current_size, dev_grid_threshold_vertex, dev_block_threshold_vertex, dev_warp_threshold_vertex);

    int host_grid_threshold_vertex;
    int host_block_threshold_vertex;
    int host_warp_threshold_vertex;
    hipMemcpy(&host_grid_threshold_vertex, dev_grid_threshold_vertex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&host_block_threshold_vertex, dev_block_threshold_vertex, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&host_warp_threshold_vertex, dev_warp_threshold_vertex, sizeof(int), hipMemcpyDeviceToHost);

    _grid_threshold_start   = 0;
    _grid_threshold_end     = host_grid_threshold_vertex;
    _block_threshold_start  = _grid_threshold_end;
    _block_threshold_end    = host_block_threshold_vertex;
    _warp_threshold_start   = _block_threshold_end;
    _warp_threshold_end     = host_warp_threshold_vertex;
    _thread_threshold_start = _warp_threshold_end;
    _thread_threshold_end   = current_size;

    hipFree(dev_grid_threshold_vertex);
    hipFree(dev_block_threshold_vertex);
    hipFree(dev_warp_threshold_vertex);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////