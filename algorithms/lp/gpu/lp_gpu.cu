#include "hip/hip_runtime.h"
#pragma once
#define REDUCE_INITIAL -1
#define DECISION_BOUND 0.2

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../architectures.h"
#define INT_ELEMENTS_PER_EDGE 3.0

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define LP_BOUNDARY_ACTIVE 1
#define LP_BOUNDARY_PASSIVE 2
#define LP_INNER 3

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../lp_constants.h"
#include "../../../graph_processing_API/gpu/cuda_API_include.h"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_segsort.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/memory.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_segreduce.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_scan.hxx"
#include "active_conditions.cuh"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Puts a 1 in differences[i] if a label of next vertice is different from its label. 0 for cases with the same labels
__global__ void label_differences_advanced(int *differences, int *dest_labels, int edges_count)
{
    long long int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((differences[i] != 1) && (i < edges_count))
    {
        if (dest_labels[i] != dest_labels[i + 1])
        {
            differences[i] = 1;
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Get rid of repeated labels by setting indices to reduce_scan array
// i-th element of reduced_scan contains index of last entry of i element in scanned array
__global__ void count_labels(int *scanned_array, long long int edges_count, int *reduced_scan)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i < edges_count) && (scanned_array[i + 1] != scanned_array[i]))
    {
        reduced_scan[scanned_array[i]] = i;
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Getting frequency of each label in reduced_scan
__global__ void frequency_count(int *frequencies, int *reduced_scan, long long int reduced_size)
{
    long long int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < reduced_size)
    {
        if ((i > 0))
        {
            frequencies[i] = reduced_scan[i] - reduced_scan[i - 1];
        }
        else
        {
            frequencies[0] = reduced_scan[0] + 1;
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Update labels
__global__ void get_labels(int *reduce_result, int *reduced_scan, int *gathered_labels, int *_labels, int vertices_count, int *_updated)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < vertices_count)
    {
        if((reduce_result[i] != -1) && (_labels[i] != gathered_labels[reduced_scan[reduce_result[i]]]))
        {
            _labels[i] = gathered_labels[reduced_scan[reduce_result[i]]];
            _updated[0] = 1;
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//Service inidices to iterate over frequencies_array
__global__ void fill_indices(int *seg_reduce_indices, long long edges_count)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < edges_count)
    {
        seg_reduce_indices[i] = i;
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void print_active_percentages(int *_node_states, int _vertices_count)
{
    int boundary_active_count = 0;
    int boundary_passive_count = 0;
    int inner_count = 0;
    for(int i = 0; i < _vertices_count; i++)
    {
        if(_node_states[i] == LP_INNER)
            inner_count++;
        if(_node_states[i] == LP_BOUNDARY_ACTIVE)
            boundary_active_count++;
        if(_node_states[i] == LP_BOUNDARY_PASSIVE)
            boundary_passive_count++;
    }

    cout << 100.0 * boundary_active_count / _vertices_count << " % boundary active" << endl;
    cout << 100.0 * boundary_passive_count / _vertices_count << " % boundary passive" << endl;
    cout << 100.0 * inner_count / _vertices_count << " % boundary inner" << endl << endl;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void print_active_ids(int *_node_states, int *_shifts, long long *_vertex_pointers, int _vertices_count)
{
    if(_vertices_count < 30)
    {
        cout << "ids of active vertices: " << endl;
        for (int src_id = 0; src_id < _vertices_count; src_id++)
        {
            if (_node_states[src_id] == LP_BOUNDARY_ACTIVE)
                printf("id=%d(CC=%ld, shift=%d) ", src_id, _vertex_pointers[src_id + 1] - _vertex_pointers[src_id], _shifts[src_id]);
        }
        cout << endl;
    }
}

template <typename _T>
void print_data(string _name, _T *_data, int _size)
{
    cout << _name << ": ";
    for(int i = 0; i < _size; i++)
    {
        cout << _data[i] << " ";
    }
    cout << endl << endl;
}

template <typename DataType, typename SegmentType>
void print_segmented_array(string _name, DataType *_data, SegmentType *_segments, int _segment_count, int _data_size)
{
    cout << _name << ": ";
    for(int i = 0; i < _data_size; i++)
    {
        cout << _data[i] << " ";
    }
    cout << endl;

    cout << _name << " with segments: ";

    for(int segment = 0; segment < _segment_count; segment++)
    {
        int start = _segments[segment];
        int end = _segments[segment + 1];
        cout << " (" << segment << ")" << "[";
        for(int i = start; i < end; i++)
        {
            if(i != (end - 1))
                cout << _data[i] << " ";
            else
                cout << _data[i];
        }
        cout << "] ";
    }
    cout << endl << endl;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename _TVertexValue, typename _TEdgeWeight>
void gpu_lp_wrapper(UndirectedCSRGraph &_graph,
                    int *_labels,
                    int &_iterations_count,
                    GpuActiveConditionType _gpu_active_condition_type,
                    int _max_iterations)
{
    LOAD_UNDIRECTED_CSR_GRAPH_DATA(_graph);
    GraphPrimitivesGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());

    mgpu::standard_context_t context;

    int *gathered_labels;
    int *tmp_work_buffer_for_seg_sort;
    int *new_ptr;
    int *label_differences;
    int *scanned;
    int *array_1;
    int *array_2;
    int *seg_reduce_indices;
    int *seg_reduce_result;
    int *reduced_scan;
    int *frequencies;
    int *node_states;
    int *shifts;
    MemoryAPI::allocate_device_array(&new_ptr, vertices_count + 1);
    MemoryAPI::allocate_device_array(&array_1, edges_count + 1);
    MemoryAPI::allocate_device_array(&array_2, edges_count +1);
    MemoryAPI::allocate_device_array(&seg_reduce_indices, edges_count + 1);
    MemoryAPI::allocate_device_array(&seg_reduce_result, vertices_count);
    MemoryAPI::allocate_device_array(&gathered_labels, edges_count + 1);
    MemoryAPI::allocate_device_array(&node_states, vertices_count);
    MemoryAPI::allocate_device_array(&shifts, vertices_count + 1);

    frontier.set_all_active();

    // init labels and node states
    if(_gpu_active_condition_type == LabelChangedRecently)
    {
        auto init_op = [_labels, node_states] __device__(int src_id, int position_in_frontier, int connections_count)
        {
            _labels[src_id] = src_id;
            node_states[src_id] = 4;
        };
        graph_API.compute(_graph, frontier, init_op);
    }
    else
    {
        auto init_op = [_labels, node_states] __device__(int src_id, int position_in_frontier, int connections_count)
        {
            _labels[src_id] = src_id;
            node_states[src_id] = LP_BOUNDARY_ACTIVE;
        };
        graph_API.compute(_graph, frontier, init_op);
    }

    _iterations_count = 0;

    int *updated;
    hipMallocManaged((void**)&updated,  sizeof(int));

    do
    {
        updated[0] = 0;

        // generate new frontier with only active nodes
        if(_gpu_active_condition_type == LabelChangedRecently)
        {
            auto node_is_active = [node_states] __device__ (int src_id)->int
            {
                if(node_states[src_id] > 0)
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };
            graph_API.generate_new_frontier(_graph, frontier, node_is_active);
        }
        else if(_gpu_active_condition_type == AlwaysActive)
        {
            frontier.set_all_active();
        }
        else
        {
            auto node_is_active = [node_states] __device__ (int src_id)->int
            {
                if(node_states[src_id] == LP_BOUNDARY_ACTIVE)
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };
            graph_API.generate_new_frontier(_graph, frontier, node_is_active);
        }
        //print_active_percentages(node_states, vertices_count); // debug part

        // calculate shifts for gathered labels
        hipMemset(shifts, 0, vertices_count*sizeof(int));
        int frontier_size = frontier.size();
        auto copy_degrees = [shifts, frontier_size] __device__(int src_id, int position_in_frontier, int connections_count)
        {
            shifts[position_in_frontier] = connections_count;
        };

        graph_API.compute(_graph, frontier, copy_degrees);
        thrust::exclusive_scan(thrust::device, shifts, shifts + frontier.size() + 1, shifts);

        auto gather_edge_op = [_labels, gathered_labels, shifts] __device__(int src_id, int dst_id, int local_edge_pos, long long int global_edge_pos, int position_in_frontier)
        {
            int dst_label = __ldg(&_labels[dst_id]);
            int src_label = _labels[src_id];
            gathered_labels[shifts[position_in_frontier] + local_edge_pos] = dst_label;
        };

        //Gathering labels of adjacent vertices
        graph_API.advance(_graph, frontier, gather_edge_op);

        int new_vertices_count = frontier.size();
        int new_edges_count = 0;
        SAFE_CALL(hipMemcpy(&new_edges_count, shifts + new_vertices_count , sizeof(int), hipMemcpyDeviceToHost));

        dim3 block_edges(1024);
        dim3 grid_edges((new_edges_count - 1) / block_edges.x + 1);

        if(new_edges_count == 0)
        {
            _iterations_count++;
            continue;
        }

        SAFE_KERNEL_CALL((fill_indices<<<grid_edges, block_edges>>>(seg_reduce_indices, new_edges_count)));

        //Sorting labels of adjacent vertices in per-vertice components.
        tmp_work_buffer_for_seg_sort = array_1;

        mgpu::segmented_sort(gathered_labels, tmp_work_buffer_for_seg_sort, new_edges_count, shifts, new_vertices_count,
                             mgpu::less_t<int>(), context);

        label_differences = array_2;

        //Puts a 1 in the last element of each segment in boundaries_array. Segments are passed by v_array
        auto label_differences_initial_op = [shifts, label_differences] __device__(int src_id, int position_in_frontier, int connections_count)
        {
            long int position = shifts[position_in_frontier];
            if(position_in_frontier != 0)
            {
                label_differences[position - 1] = 1;
            }
        };
        SAFE_CALL((hipMemset(label_differences, 0, (size_t)(sizeof(int)) * new_edges_count))); // set zeroes to difference array
        graph_API.compute(_graph, frontier, label_differences_initial_op);

        SAFE_KERNEL_CALL((label_differences_advanced <<< grid_edges, block_edges >>>
                                (label_differences, gathered_labels, new_edges_count)));

        scanned = array_1;
        //exclusive scan in order to pass repeated labels and divide different labels
        thrust::exclusive_scan(thrust::device, label_differences, label_differences + new_edges_count + 1, scanned, 0);

        int reduced_size = 0;
        SAFE_CALL(hipMemcpy(&reduced_size, scanned + new_edges_count , sizeof(int), hipMemcpyDeviceToHost));

        reduced_scan = array_2;
        SAFE_KERNEL_CALL((count_labels <<< grid_edges, block_edges >>> (scanned, new_edges_count, reduced_scan)));

        //new_ptr array contains new bounds of segments by getting them from scan
        //This is necessary due to shortened size of reduced_scan
        auto new_boundaries_op = [shifts, scanned, new_ptr] __device__(int src_id, int position_in_frontier, int connections_count)
        {
            new_ptr[position_in_frontier] = scanned[shifts[position_in_frontier]];
        };

        int position = 0;
        SAFE_CALL(hipMemcpy(&position, shifts + new_vertices_count , sizeof(int), hipMemcpyDeviceToHost));
        SAFE_CALL(hipMemcpy(new_ptr + new_vertices_count, scanned + position, sizeof(int), hipMemcpyDeviceToDevice));
        //new_ptr[new_vertices_count] = scanned[shifts[new_vertices_count]]; // TODO FIX without managed
        graph_API.compute(_graph, frontier, new_boundaries_op);

        frequencies = array_1;
        SAFE_KERNEL_CALL((frequency_count <<< grid_edges, block_edges >>> (frequencies, reduced_scan, reduced_size)));

        int init = REDUCE_INITIAL;
        auto seg_reduce_op =[frequencies, reduced_size] MGPU_DEVICE(int a, int b) ->int
        {
            int w_a = -1;
            int w_b = -1;
            if(a >= 0)
                w_a = frequencies[a];
            if(b >= 0)
                w_b = frequencies[b];

            if (w_a > w_b)
            {
                return a;
            }
            else
            {
                return b;
            }
        };

        //Searching for maximum frequency in each per-vertice segment
        mgpu::segreduce(seg_reduce_indices, reduced_size, new_ptr, new_vertices_count, seg_reduce_result,
                        seg_reduce_op, (int) init, context);

        if(_gpu_active_condition_type == AlwaysActive)
        {
            always_active(_graph, graph_API, frontier, new_ptr, gathered_labels,
                          _labels, updated, node_states, seg_reduce_result,
                          reduced_scan, seg_reduce_indices, _iterations_count);
        }
        else if(_gpu_active_condition_type == ActivePassiveInner)
        {
            active_passive_inner(_graph, graph_API, frontier, new_ptr, gathered_labels,
                                 _labels, updated, node_states, seg_reduce_result,
                                 reduced_scan, seg_reduce_indices, _iterations_count);
        }
        else if(_gpu_active_condition_type == LabelChangedOnPreviousIteration)
        {
            label_changed_on_previous_iteration(_graph, graph_API, frontier, new_ptr, gathered_labels,
                                                _labels, updated, node_states, seg_reduce_result,
                                                reduced_scan, seg_reduce_indices, _iterations_count);
        }
        else if(_gpu_active_condition_type == LabelChangedRecently)
        {
            label_changed_recently(_graph, graph_API, frontier, new_ptr, gathered_labels,
                                   _labels, updated, node_states, seg_reduce_result,
                                   reduced_scan, seg_reduce_indices, _iterations_count);
        }
        
        _iterations_count++;
        updated[0] = 1;
    }
    while((_iterations_count < _max_iterations) && (updated[0] > 0));

    MemoryAPI::free_device_array(new_ptr);
    MemoryAPI::free_device_array(array_1);
    MemoryAPI::free_device_array(array_2);
    MemoryAPI::free_device_array(seg_reduce_indices);
    MemoryAPI::free_device_array(seg_reduce_result);
    MemoryAPI::free_device_array(gathered_labels);
    MemoryAPI::free_device_array(node_states);
    MemoryAPI::free_device_array(shifts);

    hipFree(updated);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template void gpu_lp_wrapper<int, float>(UndirectedCSRGraph<int, float> &_graph, int *_labels, int &_iterations_count,
                                         GpuActiveConditionType _gpu_active_condition_type, int _max_iterations);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
