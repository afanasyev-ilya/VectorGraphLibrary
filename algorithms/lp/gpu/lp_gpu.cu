#include "hip/hip_runtime.h"
#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../graph_processing_API/gpu/cuda_API_include.h"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_segsort.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/memory.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_segreduce.hxx"
#include "../../../external_libraries/moderngpu/src/moderngpu/kernel_scan.hxx"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//Device functions
__global__ void extract_boundaries_initial(int *boundaries, long long int *v_array, int edges_count) {

    long int i = threadIdx.x + blockIdx.x * blockDim.x;
    long int position = v_array[i];
    if (i != 0) {
        boundaries[position - 1] = 1;
    } else {
        boundaries[edges_count - 1] = 1;
    }
}

__global__ void extract_boundaries_optional(int *boundaries, int *dest_labels, int edges_count) {
    long int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((boundaries[i] != 1) && (i < edges_count)) {
        if (dest_labels[i] != dest_labels[i + 1]) {
            boundaries[i] = 1;
        }
    }
}

__global__ void count_labels(int *scanned_array, int edges_count, int *S_array) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i < edges_count - 1) && (scanned_array[i + 1] != scanned_array[i])) {
        S_array[scanned_array[i]] = i;
    }
}

__global__ void new_boundaries(int *scanned_array, long long int *v_array, int edges_count, int *S_ptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    S_ptr[i] = scanned_array[v_array[i]];
}


__global__ void frequency_count(int *W_array, int *S) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i > 0) && (S[i] != 0)) {
        W_array[i] = S[i] - S[i - 1];
    } else {
        W_array[0] = S[0] + 1;
    }
}

__global__ void get_labels(int *I, int *S, int *L, int *_labels) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    _labels[i] = L[S[I[i]]];
}

__global__ void print_scanned_array(int *scanned,int *ptr, int edges_count) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    printf("%d----%d\n", i , scanned[i]);
    printf("%d----%d\n", i , ptr[i]);
}

__global__ void fill_indices(int *I) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    I[i] = i;

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

using namespace std;

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename _TVertexValue, typename _TEdgeWeight>
void gpu_lp_wrapper(ExtendedCSRGraph<_TVertexValue, _TEdgeWeight> &_graph,
                    int *_labels,
                    int &_iterations_count) {
    LOAD_EXTENDED_CSR_GRAPH_DATA(_graph);
    GraphPrimitivesGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());


    int *gathered_labels;
    int *values;
    int *s_ptr_array;
    int *F_mem;
    int *F_scanned;
    SAFE_CALL((hipMalloc((void **) &F_mem, (size_t)(sizeof(int)) * edges_count)));
    SAFE_CALL((hipMalloc((void **) &values, (size_t)(sizeof(int)) * edges_count)));
    SAFE_CALL((hipMalloc((void **) &s_ptr_array, (size_t)(sizeof(int)) * vertices_count)));
    SAFE_CALL((hipMalloc((void **) &F_scanned, (size_t)(sizeof(int)) * edges_count)));
    mgpu::standard_context_t context;
    mgpu::mem_t<int> out(vertices_count, context);
    mgpu::mem_t<int> I_mem(edges_count, context);
    MemoryAPI::allocate_device_array(&gathered_labels, edges_count);

    frontier.set_all_active();

    auto init_op =[_labels] __device__(int
    src_id, int
    connections_count) {
        _labels[src_id] = src_id;
    };

    graph_API.compute(_graph, frontier, init_op);

    _iterations_count = 0;
    while (_iterations_count < 1) // for example we can do only 1 iteration
    {
        auto gather_edge_op = [_labels, gathered_labels]
                __device__(int
        src_id, int
        dst_id, int
        local_edge_pos, long long int
        global_edge_pos){
            int dst_label = __ldg(&_labels[dst_id]);
            gathered_labels[global_edge_pos] = dst_label;
        };
        graph_API.advance(_graph, frontier, gather_edge_op);

        mgpu::segmented_sort(gathered_labels, values, edges_count, outgoing_ptrs, vertices_count,
                             mgpu::less_t<int>(), context);
        {
            dim3 block(edges_count, 1);
            dim3 grid(1, 1);
            SAFE_KERNEL_CALL(
                    (print_scanned_array<<<grid,block>>>(gathered_labels,outgoing_ptrs,edges_count))); //fill 1 in bounds
        }

        SAFE_CALL((hipMemset(F_mem, 0, (size_t)(sizeof(int)) * edges_count))); //was taken from group of memcpy
        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);
            SAFE_KERNEL_CALL(
                    (extract_boundaries_initial << < grid, block >> >
                                                           (F_mem, outgoing_ptrs, edges_count))); //fill 1 in bounds
        }
        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);

            SAFE_KERNEL_CALL(
                    (extract_boundaries_optional << < grid, block >> >
                                                            (F_mem, gathered_labels, edges_count))); //sub(i+1, i)
        }

        mgpu::scan(F_mem, edges_count, F_scanned, context); // may not work because of bool
        int reduced_size = 0;
        int *scanned_data_ptr = F_scanned;
        hipMemcpy(&reduced_size, scanned_data_ptr + (edges_count - 1), sizeof(int), hipMemcpyDeviceToHost);
        mgpu::mem_t<int> s_array(reduced_size, context);
        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);
            SAFE_KERNEL_CALL(
                    (count_labels << < grid, block >> > (F_scanned, edges_count, s_array.data())));
        }
        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);
            SAFE_KERNEL_CALL((new_boundaries << < grid, block >> >
                                                        (F_scanned, outgoing_ptrs, edges_count, s_ptr_array)));
        }
        mgpu::mem_t<int> w_array(reduced_size, context);
        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);


            SAFE_KERNEL_CALL((frequency_count << < grid, block >> > (w_array.data(), s_array.data())));
        }

        int init = 0;
        int *w_ptr = w_array.data();

        auto my_cool_lambda =[w_ptr] MGPU_DEVICE(int
        a, int
        b) ->int{
                if ( w_ptr[a] > w_ptr[b]){
                    return a;
                } else{
                    return b;
                }
        };


        mgpu::segreduce(I_mem.data(), reduced_size, s_ptr_array, vertices_count, out.data(),
                        my_cool_lambda, (int) init, context);

        {
            dim3 block(1024, 1);
            dim3 grid((edges_count - 1) / block.x + 1, 1);
            SAFE_KERNEL_CALL((get_labels << < grid, block >> >
                                                    (out.data(), s_array.data(), gathered_labels, _labels)));
        }
        _iterations_count++;
    }
    SAFE_CALL(hipFree(F_mem));
    SAFE_CALL(hipFree(values));
    SAFE_CALL(hipFree(F_scanned));
    SAFE_CALL(hipFree(s_ptr_array));


    MemoryAPI::free_device_array(gathered_labels);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template void gpu_lp_wrapper<int, float>(ExtendedCSRGraph<int, float> &_graph, int *_labels, int &_iterations_count);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
