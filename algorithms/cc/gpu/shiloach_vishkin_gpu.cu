#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../architectures.h"
#include "../../../graph_processing_API/gpu/cuda_API_include.h"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void shiloach_vishkin_wrapper(UndirectedCSRGraph &_graph,
                              int *_components,
                              int &_iterations_count)
{
    LOAD_UNDIRECTED_CSR_GRAPH_DATA(_graph);

    GraphAbstractionsGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());
    frontier.set_all_active();

    auto init_components_op = [_components] __device__ (int src_id, int position_in_frontier, int connections_count)
    {
        _components[src_id] = src_id;
    };
    graph_API.compute(_graph, frontier, init_components_op);

    int *hook_changes, *jump_changes;
    hipMallocManaged(&hook_changes, sizeof(int));
    hipMallocManaged(&jump_changes, sizeof(int));

    _iterations_count = 0;
    do
    {
        hook_changes[0] = 0;

        auto edge_op = [_components, hook_changes, _iterations_count] __device__(int src_id, int dst_id, int local_edge_pos, long long int global_edge_pos, int position_in_frontier)
        {
            int src_val = _components[src_id];
            int dst_val = _components[dst_id];

            if(src_val < dst_val)
            {
                _components[dst_id] = src_val;
                hook_changes[0] = 1;
            }

            if(src_val > dst_val)
            {
                _components[src_id] = dst_val;
                hook_changes[0] = 1;
            }
        };

        graph_API.advance(_graph, frontier, edge_op);

        do
        {
            jump_changes[0] = 0;
            auto jump_op = [_components, jump_changes] __device__(int src_id, int position_in_frontier, int connections_count)
            {
                int src_label = _components[src_id];
                int parent_label = _components[src_label];

                if(src_label != parent_label)
                {
                    _components[src_id] = parent_label;
                    jump_changes[0] = 0;
                }
            };

            graph_API.compute(_graph, frontier, jump_op);
        } while(jump_changes[0] > 0);

        _iterations_count++;
    } while(hook_changes[0] > 0);

    hipFree(hook_changes);
    hipFree(jump_changes);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template void shiloach_vishkin_wrapper<int, float>(UndirectedCSRGraph<int, float> &_graph, int *_components,
                                                   int &_iterations_count);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////