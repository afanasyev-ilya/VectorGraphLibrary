#include "hip/hip_runtime.h"
#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../architectures.h"
#define INT_ELEMENTS_PER_EDGE 3.0
//#define __PRINT_API_PERFORMANCE_STATS__

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../graph_processing_API/gpu/cuda_API_include.h"
#include "../change_state/change_state.h"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

using namespace std;

#define FIRST_LEVEL_VERTEX 1
#define UNVISITED_VERTEX -1

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define VERTICES_IN_VECTOR_EXTENSION 10

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void __global__ bottom_up_kernel(const long long *_vertex_pointers,
                                 const int *_adjacent_ids,
                                 const int _vertices_count,
                                 const int *_vector_extension,
                                 int *_levels,
                                 int _current_level,
                                 int *_vis)
{
    const int src_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(src_id < _vertices_count)
    {
        if(_levels[src_id] == UNVISITED_VERTEX)
        {
            long long start = _vertex_pointers[src_id];
            long long end = _vertex_pointers[src_id + 1];
            int connections_count = end - start;

            for (int edge_pos = start; edge_pos < end; edge_pos++)
            {
                int dst_id = _adjacent_ids[edge_pos];
                if (_levels[dst_id] == _current_level)
                {
                    _levels[src_id] = _current_level + 1;
                    atomicAdd(_vis, 1);
                    break;
                }
            }

            /*bool connected = false;
            #pragma unroll
            for(int i = 0; i < VERTICES_IN_VECTOR_EXTENSION; i++)
            {
                if(i < connections_count)
                {
                    int dst_id = _vector_extension[i * _vertices_count + src_id];
                    if (_levels[dst_id] == _current_level)
                    {
                        _levels[src_id] = _current_level + 1;
                        //atomicAdd(_vis, 1);
                        connected = true;
                        break;
                    }
                }
                else
                {
                    break;
                }
            }

            if(!connected)
            {
                for (int edge_pos = start + VERTICES_IN_VECTOR_EXTENSION; edge_pos < end; edge_pos++)
                {
                    int dst_id = _adjacent_ids[edge_pos];
                    if (_levels[dst_id] == _current_level)
                    {
                        _levels[src_id] = _current_level + 1;
                        atomicAdd(_vis, 1);
                        break;
                    }
                }
            }*/
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void __global__ ve_bottom_up_kernel(const long long *_vertex_pointers,
                                    const int *_adjacent_ids,
                                    const int _vertices_count,
                                    int *_vector_extension,
                                    int *_levels,
                                    int _current_level,
                                    int *_vis)
{
    const int src_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(src_id < _vertices_count)
    {
        if(_levels[src_id] == UNVISITED_VERTEX)
        {
            long long start = _vertex_pointers[src_id];
            long long end = _vertex_pointers[src_id + 1];
            int connections_count = end - start;
            for(int i = 0; i < VERTICES_IN_VECTOR_EXTENSION; i++)
            {
                int dst_id = _vector_extension[i * _vertices_count + src_id];
                if ((i < connections_count) && (_levels[dst_id] == _current_level))
                {
                    _levels[src_id] = _current_level + 1;
                    atomicAdd(_vis, 1);
                    break;
                }
            }
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void bottom_up_step(UndirectedCSRGraph &_graph,
                    FrontierGPU &_frontier,
                    GraphPrimitivesGPU &_graph_API,
                    const long long *_vertex_pointers,
                    const int *_adjacent_ids,
                    const int _vertices_count,
                    int *_vector_extension,
                    int *_levels,
                    int _current_level,
                    int *_vis,
                    bool _use_vector_extension)
{
    bottom_up_kernel<<< (_vertices_count - 1)/BLOCK_SIZE + 1, BLOCK_SIZE >>>(_vertex_pointers, _adjacent_ids, _vertices_count, _vector_extension, _levels, _current_level, _vis);
    hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void __global__ init_vector_extension_kernel(const long long *_vertex_pointers,
                                             const int *_adjacent_ids,
                                             const int _vertices_count,
                                             int *_vector_extension)
{
    const int src_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (src_id < _vertices_count)
    {
        long long start = _vertex_pointers[src_id];
        long long end = _vertex_pointers[src_id + 1];
        int connections_count = end - start;
        for (int i = 0; i < min(connections_count, VERTICES_IN_VECTOR_EXTENSION); i++)
        {
            _vector_extension[_vertices_count * i + src_id] = _adjacent_ids[start + i];
        }
    }
}

void init_vector_extension(const long long *_vertex_pointers,
                           const int *_adjacent_ids,
                           const int _vertices_count,
                           int *_vector_extension)
{
    init_vector_extension_kernel<<< (_vertices_count - 1)/BLOCK_SIZE + 1, BLOCK_SIZE >>>(_vertex_pointers, _adjacent_ids, _vertices_count, _vector_extension);
    hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void direction_optimizing_wrapper(UndirectedCSRGraph &_graph,
                                  int *_levels,
                                  int _source_vertex, int &_iterations_count)
{
    LOAD_UNDIRECTED_CSR_GRAPH_DATA(_graph);
    GraphPrimitivesGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());

    // unroll here - 4-5x
    int *vector_extension;
    /*MemoryAPI::allocate_device_array(&vector_extension, vertices_count * VERTICES_IN_VECTOR_EXTENSION);
    init_vector_extension(_graph.get_vertex_pointers(), _graph.get_adjacent_ids(), vertices_count, vector_extension);*/

    int *next_frontier_size;
    MemoryAPI::allocate_managed_array(&next_frontier_size, 1);

    auto init_levels = [_levels, _source_vertex] __device__ (int src_id, int position_in_frontier, int connections_count)
    {
        if(src_id == _source_vertex)
            _levels[_source_vertex] = FIRST_LEVEL_VERTEX;
        else
            _levels[src_id] = UNVISITED_VERTEX;
    };
    frontier.set_all_active();
    graph_API.compute(_graph, frontier, init_levels);

    frontier.clear();
    frontier.add_vertex(_graph, _source_vertex);

    int current_level = FIRST_LEVEL_VERTEX;
    StateOfBFS current_state = TOP_DOWN;

    int *vis;
    int *in_lvl;
    MemoryAPI::allocate_managed_array(&vis, 1);
    MemoryAPI::allocate_managed_array(&in_lvl, 1);
    vis[0] = 1;
    in_lvl[0] = 0;

    int current_frontier_size = 1, prev_frontier_size = 0;
    double t_begin = omp_get_wtime();
    do
    {
        double t1, t2;
        vis[0] = 0;

        auto reduce_op = [] __device__(int src_id, int position_in_frontier, int connections_count)->int
        {
            return connections_count;
        };
        in_lvl[0] = graph_API.reduce<int>(_graph, frontier, reduce_op, REDUCE_SUM);

        MemoryAPI::prefetch_managed_array(vis, 1);

        if(current_state == TOP_DOWN)
        {
            auto edge_op = [_levels, current_level, vis, in_lvl] __device__ (int src_id, int dst_id, int local_edge_pos, long long int global_edge_pos, int frontier_pos)
            {
                if(_levels[dst_id] == UNVISITED_VERTEX)
                {
                    _levels[dst_id] = current_level + 1;
                    atomicAdd(vis, 1);
                }
            };

            auto EMPTY_VERTEX_OP = [] __device__(int src_id, int position_in_frontier, int connections_count){};

            auto on_next_level = [_levels, current_level] __device__ (int src_id)->int
            {
                if(_levels[src_id] == (current_level + 1))
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };

            //graph_API.advance(_graph, frontier, edge_op, EMPTY_VERTEX_OP, EMPTY_VERTEX_OP, frontier, on_next_level);
            graph_API.advance(_graph, frontier, edge_op);
        }
        else if(current_state == BOTTOM_UP)
        {
            bool _use_vector_extension = false;
            //_use_vector_extension = true;
            bottom_up_step(_graph, frontier, graph_API, _graph.get_vertex_pointers(), _graph.get_adjacent_ids(), vertices_count, vector_extension, _levels, current_level, vis, _use_vector_extension);
        }

        if(vis[0] == 0)
        {
            break;
        }

        StateOfBFS next_state = gpu_change_state(current_frontier_size, vis[0], vertices_count, edges_count,
                current_state, vis[0], in_lvl[0], current_level, POWER_LAW_GRAPH);

        if(next_state == TOP_DOWN)
        {
            auto on_next_level = [_levels, current_level] __device__ (int src_id)->int
            {
                if(_levels[src_id] == (current_level + 1))
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };
            graph_API.generate_new_frontier(_graph, frontier, on_next_level);
        }

        current_state = next_state;
        current_frontier_size = vis[0];
        prev_frontier_size = current_frontier_size;
        current_level++;

    } while(vis[0] > 0);
    double t_end = omp_get_wtime();
    //cout << "inner perf: " << edges_count / ((t_end - t_begin)*1e6) << " TEPS" << endl;

    _iterations_count = current_level;

    //MemoryAPI::free_device_array(vector_extension);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template void direction_optimizing_wrapper<int, float>(UndirectedCSRGraph<int, float> &_graph, int *_levels,
                                                       int _source_vertex, int &_iterations_count);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

