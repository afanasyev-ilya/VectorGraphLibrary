#include "hip/hip_runtime.h"
#pragma once

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define INT_ELEMENTS_PER_EDGE 3.0

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "../../../graph_processing_API/gpu/cuda_API_include.h"
#include "../change_state/change_state.h"

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

using namespace std;

#define FIRST_LEVEL_VERTEX 1
#define UNVISITED_VERTEX -1

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename _TVertexValue, typename _TEdgeWeight>
void top_down_wrapper(ExtendedCSRGraph<_TVertexValue, _TEdgeWeight> &_graph,
                      int *_levels,
                      int _source_vertex,
                      int &_iterations_count)
{
    LOAD_EXTENDED_CSR_GRAPH_DATA(_graph);
    GraphPrimitivesGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());

    frontier.set_all_active();

    auto init_levels = [_levels, _source_vertex] __device__ (int src_id, int position_in_frontier, int connections_count)
    {
        if(src_id == _source_vertex)
            _levels[_source_vertex] = FIRST_LEVEL_VERTEX;
        else
            _levels[src_id] = UNVISITED_VERTEX;
    };
    graph_API.compute(_graph, frontier, init_levels);

    auto on_first_level = [_levels] __device__ (int src_id)->int
    {
        if(_levels[src_id] == FIRST_LEVEL_VERTEX)
            return IN_FRONTIER_FLAG;
        else
            return NOT_IN_FRONTIER_FLAG;
    };
    graph_API.generate_new_frontier(_graph, frontier, on_first_level);

    int current_level = FIRST_LEVEL_VERTEX;
    while(frontier.size() > 0)
    {
        auto edge_op = [_levels, current_level] __device__ (int src_id, int dst_id, int local_edge_pos, long long int global_edge_pos, int frontier_pos)
        {
            int src_level = _levels[src_id];
            int dst_level = _levels[dst_id];
            if((src_level == current_level) && (dst_level == UNVISITED_VERTEX))
            {
                _levels[dst_id] = current_level + 1;
            }
        };

        graph_API.advance(_graph, frontier, edge_op);

        auto on_next_level = [_levels, current_level] __device__ (int src_id)->int
        {
            if(_levels[src_id] == (current_level + 1))
                return IN_FRONTIER_FLAG;
            else
                return NOT_IN_FRONTIER_FLAG;
        };

        graph_API.generate_new_frontier(_graph, frontier, on_next_level);

        current_level++;
    }
    _iterations_count = current_level;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void __global__ bottom_up_kernel(const long long *_vertex_pointers,
                                 const int *_adjacent_ids,
                                 const int _vertices_count,
                                 int *_levels,
                                 int _current_level,
                                 int *_vis)
{
    const int src_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(src_id < _vertices_count)
    {
        if(_levels[src_id] == UNVISITED_VERTEX)
        {
            long long start = _vertex_pointers[src_id];
            long long end = _vertex_pointers[src_id + 1];
            int connections_count = end - start;
            for (int edge_pos = start; edge_pos < end; edge_pos++)
            {
                int dst_id = _adjacent_ids[edge_pos];
                if (_levels[dst_id] == _current_level)
                {
                    _levels[src_id] = _current_level + 1;
                    atomicAdd(_vis, 1);
                    break;
                }
            }
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void bottom_up_step(const long long *_vertex_pointers,
                    const int *_adjacent_ids,
                    const int _vertices_count,
                    int *_levels,
                    int _current_level,
                    int *_vis)
{
    bottom_up_kernel<<< (_vertices_count - 1)/BLOCK_SIZE + 1, BLOCK_SIZE >>>(_vertex_pointers, _adjacent_ids, _vertices_count, _levels, _current_level, _vis);
    hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename _TVertexValue, typename _TEdgeWeight>
void direction_optimizing_wrapper(ExtendedCSRGraph<_TVertexValue, _TEdgeWeight> &_graph,
                                  int *_levels,
                                  int _source_vertex, int &_iterations_count)
{
    LOAD_EXTENDED_CSR_GRAPH_DATA(_graph);
    GraphPrimitivesGPU graph_API;
    FrontierGPU frontier(_graph.get_vertices_count());

    int *next_frontier_size;
    MemoryAPI::allocate_managed_array(&next_frontier_size, 1);

    auto init_levels = [_levels, _source_vertex] __device__ (int src_id, int position_in_frontier, int connections_count)
    {
        if(src_id == _source_vertex)
            _levels[_source_vertex] = FIRST_LEVEL_VERTEX;
        else
            _levels[src_id] = UNVISITED_VERTEX;
    };
    frontier.set_all_active();
    graph_API.compute(_graph, frontier, init_levels);

    frontier.clear();
    frontier.add_vertex(_graph, _source_vertex);

    int current_level = FIRST_LEVEL_VERTEX;
    StateOfBFS current_state = TOP_DOWN;

    int *vis;
    int *in_lvl;
    MemoryAPI::allocate_managed_array(&vis, 1);
    MemoryAPI::allocate_managed_array(&in_lvl, 1);
    vis[0] = 1;
    in_lvl[0] = 0;

    double total_time = 0;
    double reduce_time = 0, advance_time = 0, gnf_time = 0;

    int current_frontier_size = 1, prev_frontier_size = 0;
    double t_begin = omp_get_wtime();
    do
    {
        double t1, t2;
        cout << "state: " << current_state << endl;
        cout << "prev_frontier_size: " << prev_frontier_size << endl;
        vis[0] = 0;

        t1 = omp_get_wtime();
        auto reduce_op = [] __device__ (int src_id, int connections_count)->int
        {
            return connections_count;
        };
        in_lvl[0] = graph_API.reduce<int>(_graph, frontier, reduce_op, REDUCE_SUM);
        t2 = omp_get_wtime();
        reduce_time += t2 - t1;
        cout << "reduces time: " << 1000.0*(t2 - t1) << " ms" << endl;
        cout << "in lvl (for estimating next front): " << in_lvl[0] << " " << 100.0*in_lvl[0]/vertices_count << endl;

        MemoryAPI::prefetch_managed_array(vis, 1);

        t1 = omp_get_wtime();
        if(current_state == TOP_DOWN)
        {
            auto edge_op = [_levels, current_level, vis, in_lvl] __device__ (int src_id, int dst_id, int local_edge_pos, long long int global_edge_pos, int frontier_pos)
            {
                if(_levels[dst_id] == UNVISITED_VERTEX)
                {
                    _levels[dst_id] = current_level + 1;
                    atomicAdd(vis, 1);
                }
            };

            auto EMPTY_VERTEX_OP = [] __device__(int src_id, int position_in_frontier, int connections_count){};

            auto on_next_level = [_levels, current_level] __device__ (int src_id)->int
            {
                if(_levels[src_id] == (current_level + 1))
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };

            graph_API.advance(_graph, frontier, edge_op, EMPTY_VERTEX_OP, EMPTY_VERTEX_OP, frontier, on_next_level);
        }
        else if(current_state == BOTTOM_UP)
        {
            bottom_up_step(_graph.get_outgoing_ptrs(), _graph.get_outgoing_ids(), vertices_count, _levels, current_level, vis);
        }
        t2 = omp_get_wtime();
        cout << "td/bu time: " << 1000.0*(t2 - t1) << " ms" << endl;
        total_time += t2 - t1;
        advance_time += t2 - t1;

        if(vis[0] == 0)
        {
            break;
        }

        StateOfBFS next_state = gpu_change_state(current_frontier_size, vis[0], vertices_count, edges_count,
                current_state, vis[0], in_lvl[0], current_level, POWER_LAW_GRAPH);

        if((current_state == BOTTOM_UP) && (next_state == TOP_DOWN))
        {
            t1 = omp_get_wtime();
            auto on_next_level = [_levels, current_level] __device__ (int src_id)->int
            {
                if(_levels[src_id] == (current_level + 1))
                    return IN_FRONTIER_FLAG;
                else
                    return NOT_IN_FRONTIER_FLAG;
            };
            graph_API.generate_new_frontier(_graph, frontier, on_next_level);
            t2 = omp_get_wtime();
            cout << "gnf time: " << 1000.0*(t2 - t1) << " ms" << endl;
            gnf_time += t2 - t1;
        }

        current_state = next_state;
        current_frontier_size = vis[0];
        prev_frontier_size = current_frontier_size;
        current_level++;

        cout << "vis: " << vis[0] << endl;
        cout << endl;
    } while(vis[0] > 0);
    double t_end = omp_get_wtime();
    cout << "inner time: " << total_time*1000 << " ms" <<  endl;
    cout << "inner 2 time: " << 1000*(t_end - t_begin) << " ms" << endl;
    cout << "reduce_time: " << reduce_time * 1000  << " ms" << endl;
    cout << "advance_time: " << advance_time * 1000  << " ms" << endl;
    cout << "gnf_time: " << gnf_time * 1000  << " ms" << endl;

    _iterations_count = current_level;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


template void top_down_wrapper<int, float>(ExtendedCSRGraph<int, float> &_graph, int *_levels,
                                           int _source_vertex, int &_iterations_count);
template void direction_optimizing_wrapper<int, float>(ExtendedCSRGraph<int, float> &_graph, int *_levels,
                                                       int _source_vertex, int &_iterations_count);

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

